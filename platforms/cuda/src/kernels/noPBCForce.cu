#include "hip/hip_runtime.h"
extern "C" __global__ void calcNoPBCEnForces(
    mixed*              __restrict__     energyBuffer,
    const real4*        __restrict__     posq,
    unsigned long long* __restrict__     forceBuffers,
    const real*         __restrict__     charges,
    real*               __restrict__     dedq,
    const int*          __restrict__     atomIndex,
    const int*          __restrict__     pairidx0,
    const int*          __restrict__     pairidx1,
    int                                  numParticles,
    int                                  paddedNumAtoms) {
    int totpair = numParticles * (numParticles - 1) / 2;
    for (int npair = blockIdx.x*blockDim.x+threadIdx.x; npair < totpair; npair += blockDim.x*gridDim.x) {
        int ii = pairidx0[npair];
        int jj = pairidx1[npair];
        real3 delta = make_real3(posq[jj].x-posq[ii].x,posq[jj].y-posq[ii].y,posq[jj].z-posq[ii].z);
        real R2 = delta.x * delta.x + delta.y * delta.y + delta.z * delta.z;
        real inverseR = RSQRT(R2);
        real c1c2 = charges[atomIndex[ii]] * charges[atomIndex[jj]];
        atomicAdd(&energyBuffer[ii], ONE_4PI_EPS0 * c1c2 * inverseR);
        real dEdRdR = ONE_4PI_EPS0 * c1c2 * inverseR * inverseR * inverseR;
        real3 force = - dEdRdR * delta;
        atomicAdd(&forceBuffers[ii], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[ii+paddedNumAtoms], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[ii+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        atomicAdd(&forceBuffers[jj], static_cast<unsigned long long>((long long) (-force.x*0x100000000)));
        atomicAdd(&forceBuffers[jj+paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.y*0x100000000)));
        atomicAdd(&forceBuffers[jj+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.z*0x100000000)));

        atomicAdd(&dedq[atomIndex[ii]], ONE_4PI_EPS0*charges[atomIndex[jj]]*inverseR);
        atomicAdd(&dedq[atomIndex[jj]], ONE_4PI_EPS0*charges[atomIndex[ii]]*inverseR);
    }
}

extern "C" __global__ void calcNoPBCExclusions(
    mixed*              __restrict__     energyBuffer,
    const real4*        __restrict__     posq,
    unsigned long long* __restrict__     forceBuffers,
    const real*         __restrict__     charges,
    real*               __restrict__     dedq,
    const int*          __restrict__     atomIndex,
    const int*          __restrict__     expairidx0,
    const int*          __restrict__     expairidx1,
    const int                            totpair,
    const int                            numParticles,
    const int                            paddedNumAtoms) {
    for (int npair = blockIdx.x*blockDim.x+threadIdx.x; npair < totpair; npair += blockDim.x*gridDim.x) {
        int ii = expairidx0[npair];
        int jj = expairidx1[npair];
        real3 delta = make_real3(posq[jj].x-posq[ii].x,posq[jj].y-posq[ii].y,posq[jj].z-posq[ii].z);
        real R2 = delta.x * delta.x + delta.y * delta.y + delta.z * delta.z;
        real inverseR = RSQRT(R2);
        real c1c2 = charges[atomIndex[ii]] * charges[atomIndex[jj]];
        energyBuffer[npair] -= ONE_4PI_EPS0 * c1c2 * inverseR;
        real dEdRdR = ONE_4PI_EPS0 * c1c2 * inverseR * inverseR * inverseR;
        real3 force = dEdRdR * delta;
        atomicAdd(&forceBuffers[ii], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[ii+paddedNumAtoms], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[ii+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
        atomicAdd(&forceBuffers[jj], static_cast<unsigned long long>((long long) (-force.x*0x100000000)));
        atomicAdd(&forceBuffers[jj+paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.y*0x100000000)));
        atomicAdd(&forceBuffers[jj+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (-force.z*0x100000000)));

        atomicAdd(&dedq[atomIndex[ii]], -ONE_4PI_EPS0*charges[atomIndex[jj]]*inverseR);
        atomicAdd(&dedq[atomIndex[jj]], -ONE_4PI_EPS0*charges[atomIndex[ii]]*inverseR);
    }
}